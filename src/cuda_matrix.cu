#include "hip/hip_runtime.h"
#pragma once

#include <iostream>
#include <stdexcept>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "matrix.hpp"

namespace cuda_matrix
{
    template<typename T>
    __global__
    void CUDA_matrixMultiply(const T* A, const T* B, T* P, int width, int rows, int cols)
    {
        // Row major format
        int row = blockIdx.y * blockDim.y + threadIdx.y;   
        int col = blockIdx.x * blockDim.x + threadIdx.x;

        if (row < rows && col < cols)
        {
            int i = row * cols + col;
            T product = 0;
            for(int k = 0; k < width; k++)
            {
                product += A[row * width + k] * B[k * cols + col];
            }
            P[i] = product;
        }
    }

    template<typename T>
    T* matrixMultiplyFlattened(matrix<T> A, matrix<T> B, int dim_grid_x, int dim_grid_y, int dim_grid_z, int dim_block_x, int dim_block_y, int dim_block_z)
    {
        if (A.cols != B.rows)
        {
            std::cerr << "Matrix A columns (" << A.cols << ") must equal matrix B rows (" << B.rows << ")" << std::endl;
            throw std::invalid_argument("Matrix A columns and matrix B rows are not equal.");
        }

        T* P_flattened, *d_A, *d_B, *d_P;

        matrix<T> P(A.rows, B.cols);

        P_flattened = (T*) malloc(sizeof(T) * P.SIZE);
        hipMalloc(&d_A, sizeof(T) * A.SIZE);
        hipMalloc(&d_B, sizeof(T) * B.SIZE);
        hipMalloc(&d_P, sizeof(T) * P.SIZE);

        hipMemcpy(d_A, A.DATA, A.SIZE * sizeof(T), hipMemcpyHostToDevice);
        hipMemcpy(d_B, B.DATA, B.SIZE * sizeof(T), hipMemcpyHostToDevice);

        CUDA_matrixMultiply<<<dim3(dim_grid_x, dim_grid_y, dim_grid_z), dim3(dim_block_x, dim_block_y, dim_block_z)>>>(d_A, d_B, d_P, A.cols, P.rows, P.cols);

        hipDeviceSynchronize();

        hipMemcpy(P_flattened, d_P, P.SIZE * sizeof(T), hipMemcpyDeviceToHost);
        
        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_P);
        free(P);

        return P_flattened;
    }

    template<typename T>
    matrix<T> matrixMultiply(matrix<T> A, matrix<T> B, int dim_grid_x, int dim_grid_y, int dim_grid_z, int dim_block_x, int dim_block_y, int dim_block_z)
    {
        if (A.cols != B.rows)
        {
            std::cerr << "Matrix A columns (" << A.cols << ") must equal matrix B rows (" << B.rows << ")" << std::endl;
            throw std::invalid_argument("Matrix A columns and matrix B rows are not equal.");
        }

        T* d_A, *d_B, *d_P;

        matrix<T> P(A.rows, B.cols);

        hipMalloc(&d_A, sizeof(T) * A.SIZE);
        hipMalloc(&d_B, sizeof(T) * B.SIZE);
        hipMalloc(&d_P, sizeof(T) * P.SIZE);

        hipMemcpy(d_A, A.DATA, A.SIZE * sizeof(T), hipMemcpyHostToDevice);
        hipMemcpy(d_B, B.DATA, B.SIZE * sizeof(T), hipMemcpyHostToDevice);

        CUDA_matrixMultiply<<<dim3(dim_grid_x, dim_grid_y, dim_grid_z), dim3(dim_block_x, dim_block_y, dim_block_z)>>>(d_A, d_B, d_P, A.cols, P.rows, P.cols);

        hipDeviceSynchronize();

        hipMemcpy(P.DATA, d_P, P.SIZE * sizeof(T), hipMemcpyDeviceToHost);
        
        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_P);

        return P;
    }
}
